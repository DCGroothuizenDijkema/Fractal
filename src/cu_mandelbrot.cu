#include "hip/hip_runtime.h"

//+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+//
//                                                                                                                                       //
// cu_mandelbrot.cu                                                                                                                      //
//                                                                                                                                       //
// D. C. Groothuizen Dijkema - April, 2020                                                                                               //
//+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+//

// CUDA accelerated implementation for producing fractals from the Mandelbrot Set


#include <cu_fractal.hpp>

__device__ int iterate(hipDoubleComplex x, const hipDoubleComplex &c, const int max_itr)
{
  for (int itr=0;itr<max_itr;++itr)
  {
    if (hipCabs(x)>2) { return itr; }
    x=hipCadd(hipCmul(x,x),c);
  }
  return NPP_MAX_32S;
}

__global__ void compute_mandelbrot(int * const d_iterations, const int max_itr, const int xresolution, const int yresolution
  , const double startx, const double starty, const double deltax, const double deltay)
{
  // determine where we are in memory
  const int idy=blockIdx.y*blockDim.y+threadIdx.y,idx=blockIdx.x*blockDim.x+threadIdx.x,ind=idy*xresolution+idx;
  // check we haven't gone out of bounds
  if (idx>=xresolution||idy>=yresolution) { return; }

  // determine the current point
  const double imag=starty+deltay*idy,real=startx+deltax*idx;
  // determine the number of iterations
  d_iterations[ind]=iterate(make_hipDoubleComplex(0.,0.),make_hipDoubleComplex(real,imag),max_itr);
}

__global__ void compute_julia(int * const d_iterations, const hipDoubleComplex &c, const int max_itr
  , const int xresolution, const int yresolution, const double startx, const double starty, const double deltax, const double deltay)
{
  // determine where we are in memory
  const int idy=blockIdx.y*blockDim.y+threadIdx.y,idx=blockIdx.x*blockDim.x+threadIdx.x,ind=idy*xresolution+idx;
  // check we haven't gone out of bounds
  if (idx>=xresolution||idy>=yresolution) { return; }

  // determine the current point
  const double imag=starty+deltay*idy,real=startx+deltax*idx;
  // determine the number of iterations
  d_iterations[ind]=iterate(make_hipDoubleComplex(real,imag),c,max_itr);
}

int __declspec(dllexport) sample_mandelbrot(int * const h_itr, const int max_itr, const int xresolution, const int yresolution
  , const double startx, const double endx, const double starty, const double endy, const bool verbose)
{
  // computation parameters
  const double deltax=(endx-startx)/xresolution,deltay=(endy-starty)/yresolution;
  const int total=xresolution*yresolution;
  // memory parameters
  const int i_size=total*sizeof(int);

  // device memory pointers
  int *d_itr=nullptr;
  // allocate device memory
  CUDA_REQUIRE_SUCCESS(hipMalloc(reinterpret_cast<void **>(&d_itr),static_cast<size_t>(i_size)));

  // GPU memory setup
  const dim3 dim_block(32,32),dim_grid((xresolution+dim_block.x-1)/dim_block.x,(yresolution+dim_block.y-1)/dim_block.y);
  // run and time
  float elapsed;
  hipEvent_t start,stop;

  CUDA_REQUIRE_SUCCESS(hipEventCreate(&start));
  CUDA_REQUIRE_SUCCESS(hipEventCreate(&stop));
  CUDA_REQUIRE_SUCCESS(hipEventRecord(start,0));

  compute_mandelbrot<<<dim_grid,dim_block>>>(d_itr,max_itr,xresolution,yresolution,startx,starty,deltax,deltay);
  // check for errors
  CUDA_REQUIRE_SUCCESS(hipPeekAtLastError());
  CUDA_REQUIRE_SUCCESS(hipDeviceSynchronize());

  CUDA_REQUIRE_SUCCESS(hipEventRecord(stop,0));
  CUDA_REQUIRE_SUCCESS(hipEventSynchronize(stop));
  CUDA_REQUIRE_SUCCESS(hipEventElapsedTime(&elapsed,start,stop));

  if (verbose)
  {
    std::cout << total << " points processed." << std::endl
      << "Time taken: " << elapsed/1000 << "s." << std::endl;
  }

  // copy back to host
  CUDA_REQUIRE_SUCCESS(hipMemcpy(h_itr,d_itr,static_cast<size_t>(i_size),hipMemcpyDeviceToHost));

  // free GPU memory
  CUDA_REQUIRE_SUCCESS(hipFree(d_itr));

  return NPP_MAX_32S;
}

int __declspec(dllexport) sample_julia(int * const h_itr, const int max_itr, const int xresolution, const int yresolution
  , const double startx, const double endx, const double starty, const double endy, const bool verbose)
{
  return NPP_MAX_32S;
}
