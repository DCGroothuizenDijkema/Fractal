#include "hip/hip_runtime.h"

//+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+//
//                                                                                                                                       //
// cu_mandelbrot.cu                                                                                                                      //
//                                                                                                                                       //
// D. C. Groothuizen Dijkema - April, 2020                                                                                               //
//+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+//

// CUDA accelerated implementation for producing fractals from the Mandelbrot Set


#include <cu_fractal.hpp>

__device__ int iterate(hipDoubleComplex x, const hipDoubleComplex &c, const int max_itr);

__global__ void compute_mandelbrot(int * const d_iterations, const int max_itr, const int xresolution, const int yresolution
  , const double startx, const double starty, const double deltax, const double deltay, const int total, bool verbose);

int __declspec(dllexport) sample_mandelbrot(int * const h_iterations, const int max_itr, const int xresolution, const int yresolution
  , const double startx, const double endx, const double starty, const double endy, const bool verbose);
