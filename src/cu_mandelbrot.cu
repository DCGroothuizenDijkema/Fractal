#include "hip/hip_runtime.h"

//+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+//
//                                                                                                                                       //
// cu_mandelbrot.cu                                                                                                                      //
//                                                                                                                                       //
// D. C. Groothuizen Dijkema - April, 2020                                                                                               //
//+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+//

// CUDA accelerated implementation for producing fractals from the Mandelbrot Set


#include <cu_fractal.hpp>

__device__ int iterate(hipDoubleComplex x, const hipDoubleComplex &c, const int max_itr)
{
  for (int itr=0;itr<max_itr;++itr)
  {
    if (hipCabs(x)>2) { return itr; }
    x=hipCadd(hipCmul(x,x),c);
  }
  return NPP_MAX_32S;
}

__global__ void compute_mandelbrot(int * const d_iterations, const int max_itr, const int xresolution, const int yresolution
  , const double startx, const double starty, const double deltax, const double deltay, const int total, bool verbose)
{
  // determine where we are in memory
  const int idy=blockIdx.y*blockDim.y+threadIdx.y,idx=blockIdx.x*blockDim.x+threadIdx.x,ind=idy*xresolution+idx;
  // check we haven't gone out of bounds
  if (idx>=xresolution||idy>=yresolution) { return; }

  // determine the current point
  const double imag=starty+deltay*idy,real=startx+deltax*idx;
  // determine the number of iterations
  d_iterations[ind]=iterate(make_hipDoubleComplex(0.,0.),make_hipDoubleComplex(real,imag),max_itr);
}

int __declspec(dllexport) sample_mandelbrot(int * const h_iterations, const int max_itr, const int xresolution, const int yresolution
  , const double startx, const double endx, const double starty, const double endy, const bool verbose);
