#include "hip/hip_runtime.h"

//+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+//
//                                                                                                                                       //
// cu_fractal.hpp                                                                                                                        //
//                                                                                                                                       //
// D. C. Groothuizen Dijkema - April, 2020                                                                                               //
//+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+//

// CUDA accelerated implementation for producing Newton's fractals


#include <cu_fractal.hpp>

__device__ thrust::pair<hipDoubleComplex,hipDoubleComplex> polynomial_and_deriv(const hipDoubleComplex &x, const double * const coeffs
  , const int degree)
{
  hipDoubleComplex p,p_prime;
  p=make_hipDoubleComplex(*(coeffs+degree),0.);
  p_prime=make_hipDoubleComplex(0.,0.);

  for (int itr=degree-1;itr>=0;--itr)
  {
    p_prime=hipCadd(hipCmul(x,p_prime),p);
    p=hipCadd(hipCmul(x,p),make_hipDoubleComplex(*(coeffs+itr),0.0));
  }

  return thrust::make_pair(p,p_prime);
}

__device__ hipDoubleComplex newton_root(const double * const coeffs, int * const itr_taken, hipDoubleComplex x, const int degree
  , const int max_itr, const double tol)
{
  for (int itr=0;itr<max_itr;++itr)
  {
    // get the current function value and derivative
    hipDoubleComplex f_x,g_x;
    thrust::tie(f_x,g_x)=polynomial_and_deriv(x,coeffs,degree);
    // converged to a root
    if (hipCabs(f_x)<tol)
    {
      *itr_taken=itr;
      return x;
    }
    // derivative is flat and we can't update
    if (hipCreal(g_x)==0.&&hipCimag(g_x)==0.)
    {
      *itr_taken=NPP_MAX_32S ;
      return make_hipDoubleComplex(HIP_INF,HIP_INF);
    }
    // update
    x=hipCsub(x,hipCdiv(f_x,g_x));
  }
  // couldn't find a root in the given number of iterations
  *itr_taken=NPP_MAX_32S ;
  return make_hipDoubleComplex(HIP_INF,HIP_INF);
}

__global__ void compute_newton(double *d_re, double *d_im, int *d_itr, double * const coeffs, const int max_itr, const int degree
  , const int xresolution, const int yresolution, const double startx, const double starty, const double deltax, const double deltay)
{
  const int idx=blockIdx.x*blockDim.x+threadIdx.x,idy=threadIdx.y,offset=(idx*xresolution)+idy;
  // thread check
  if (idx>=xresolution||idy>=yresolution) { return; }
  const double imag=starty+deltay*idy,real=startx+deltax*idx;

  // determine the root reached and the number of iterations to get there
  hipDoubleComplex root=newton_root(coeffs,(d_itr+offset),make_hipDoubleComplex(real,imag),degree,max_itr,1e-6);
  *(d_re+offset)=hipCreal(root);
  *(d_im+offset)=hipCimag(root);
}

void __declspec(dllexport) sample_newton(double *h_re, double *h_im, int *h_itr, double *coeffs, const int max_itr, const int degree
  , const int xresolution, const int yresolution, const double startx, const double endx, const double starty, const double endy)
{
  double *d_re=nullptr,*d_im=nullptr;
  int *d_itr=nullptr;

  const double deltax=(endx-startx)/xresolution,deltay=(endy-starty)/yresolution;
  const int total=xresolution*yresolution,d_size=total*sizeof(double),i_size=total*sizeof(int);

  hipMalloc(reinterpret_cast<void **>(&d_re),static_cast<size_t>(d_size));
  hipMalloc(reinterpret_cast<void **>(&d_im),static_cast<size_t>(d_size));
  hipMalloc(reinterpret_cast<void **>(&d_itr),static_cast<size_t>(i_size));

  dim3 dim_block(32,32),dim_grid((xresolution*yresolution)/(dim_block.x*dim_block.y),1,1);

  compute_newton<<<dim_grid,dim_block>>>(d_re,d_im,d_itr,coeffs,max_itr,degree,xresolution,yresolution,startx,starty,deltax,deltay);

  hipMemcpy(h_re,d_re,static_cast<size_t>(total),hipMemcpyDeviceToHost);
  hipMemcpy(h_im,d_im,static_cast<size_t>(total),hipMemcpyDeviceToHost);
  hipMemcpy(h_itr,d_itr,static_cast<size_t>(total),hipMemcpyDeviceToHost);

  hipFree(d_re);
  hipFree(d_im);
  hipFree(d_itr);
}

void __declspec(dllexport) assign_roots(int *index, double *re, double *im, const double * const roots_re, const double * const roots_im
  , const int degree, const int xresolution, const int yresolution)
{
  // get a list of all roots, formed from the input vectors giving the real and imaginary components of the roots
  std::vector<std::complex<double>> roots;
  zip(roots_re,roots_re+degree,roots_im,roots_im+degree,std::back_inserter(roots));
  
  for (int itr=0;itr<xresolution*yresolution;++itr)
  {
    // if the current value is marked with infinity, no root was reached from it and its index is 0
    if (*(re+itr)==std::numeric_limits<double>::infinity()||*(im+itr)==std::numeric_limits<double>::infinity())
    {
      *(index+itr)=-1;
      continue;
    }

    std::complex<double> val(*(re+itr),*(im+itr));
    // determine the difference between the current value and all roots
    std::vector<std::complex<double>> diffs;
    std::transform(std::begin(roots),std::end(roots),std::back_inserter(diffs)
      ,[val](std::complex<double> root) { return abs(root-val); });
    // find the argmin of the differences, and, therefore, which root was converged to
    *(index+itr)=static_cast<int>(argmin(std::cbegin(diffs),std::cend(diffs)
      ,[](const std::complex<double> &x, const std::complex<double> &y){ return abs(x)<abs(y); }
    ));
  }
}
