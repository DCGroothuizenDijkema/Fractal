
//+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+//
//                                                                                                                                       //
// cu_fractal.hpp                                                                                                                        //
//                                                                                                                                       //
// D. C. Groothuizen Dijkema - April, 2020                                                                                               //
//+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+//

// CUDA accelerated implementation for producing Newton's fractals


#include <cu_fractal.hpp>

__device__ thrust::pair<hipDoubleComplex,hipDoubleComplex> polynomial_and_deriv(const hipDoubleComplex &x, const double * const coeffs
  , const int degree)
{
  hipDoubleComplex p,p_prime;
  p=make_hipDoubleComplex(*(coeffs+degree),0.);
  p_prime=make_hipDoubleComplex(0.,0.);

  for (int itr=degree-1;itr>=0;--itr)
  {
    p_prime=hipCadd(hipCmul(x,p_prime),p);
    p=hipCadd(hipCmul(x,p),make_hipDoubleComplex(*(coeffs+itr),0.0));
  }

  return thrust::make_pair(p,p_prime);
}

__device__ hipDoubleComplex newton_root(const double * const coeffs, int * const itr_taken, hipDoubleComplex x, const int degree
  , const int max_itr, const double tol)
{
  for (int itr=0;itr<max_itr;++itr)
  {
    // get the current function value and derivative
    hipDoubleComplex f_x,g_x;
    thrust::tie(f_x,g_x)=polynomial_and_deriv(x,coeffs,degree);
    // converged to a root
    if (hipCabs(f_x)<tol)
    {
      *itr_taken=itr;
      return x;
    }
    // derivative is flat and we can't update
    if (hipCreal(g_x)==0.&&hipCimag(g_x)==0.)
    {
      *itr_taken=NPP_MAX_32S ;
      return make_hipDoubleComplex(HIP_NAN,HIP_NAN);
    }
    // update
    x=hipCsub(x,hipCdiv(f_x,g_x));
  }
  // couldn't find a root in the given number of iterations
  *itr_taken=NPP_MAX_32S ;
  return make_hipDoubleComplex(HIP_NAN,HIP_NAN);
}
